#include "cuda_defines.h"
#include "cuda_stuff.h"


void CUDA_Initialize(const int rank, const int sx, const int sy, const int sz, const int bord,
               float dx, float dy, float dz, float dt)
{
extern float* vpz;
extern float* vsv;
extern float* epsilon;
extern float* delta;
extern float* phi;
extern float* theta;
extern float* ch1dxx;
extern float* ch1dyy;
extern float* ch1dzz;
extern float* ch1dxy;
extern float* ch1dyz;
extern float* ch1dxz;
extern float* v2px;
extern float* v2pz;
extern float* v2sz;
extern float* v2pn;
extern float* pp;
extern float* pc;
extern float* qp;
extern float* qc;  
extern float* fatAbsorb;

 
  // Set the device number based on rank
  int deviceCount;
  CUDA_CALL(hipGetDeviceCount(&deviceCount));
  const int device=rank%deviceCount;
  hipDeviceProp_t deviceProp;
  CUDA_CALL(hipGetDeviceProperties(&deviceProp, device));
  printf("Using device(%d) %s with compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
  CUDA_CALL(hipSetDevice(device));


  // Check sx,sy values
  if (sx%BSIZE_X != 0)
  {
     printf("sx(%d) must be multiple of BSIZE_X(%d)\n", sx, (int)BSIZE_X);
     exit(1);
  } 
  if (sy%BSIZE_Y != 0)
  {
     printf("sy(%d) must be multiple of BSIZE_Y(%d)\n", sy, (int)BSIZE_Y);
     exit(1);
  } 

   const size_t sxsysz=((size_t)sx*sy)*sz;
   const size_t msize_vol=sxsysz*sizeof(float);

   CUDA_CALL(hipMallocManaged(&vpz, msize_vol));
   //dev_vpz, vpz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&vsv, msize_vol));
   //dev_vsv, vsv, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&epsilon, msize_vol));
   //dev_epsilon, epsilon, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&delta, msize_vol));
   //dev_delta, delta, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&phi, msize_vol));
   //dev_phi, phi, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&theta, msize_vol));
   //dev_theta, theta, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&ch1dxx, msize_vol));
   //dev_ch1dxx, ch1dxx, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&ch1dyy, msize_vol));
   //dev_ch1dyy, ch1dyy, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&ch1dzz, msize_vol));
   //dev_ch1dzz, ch1dzz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&ch1dxy, msize_vol));
   //dev_ch1dxy, ch1dxy, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&ch1dyz, msize_vol));
   //dev_ch1dyz, ch1dyz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&ch1dxz, msize_vol));
   //dev_ch1dxz, ch1dxz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&v2px, msize_vol));
   //dev_v2px, v2px, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&v2pz, msize_vol));
   //dev_v2pz, v2pz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&v2sz, msize_vol));
   //dev_v2sz, v2sz, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&v2pn, msize_vol));
   //dev_v2pn, v2pn, msize_vol, hipMemcpyHostToDevice));
   CUDA_CALL(hipMallocManaged(&pp, msize_vol));
   CUDA_CALL(hipMemset(pp, 0, msize_vol));
   CUDA_CALL(hipMallocManaged(&pc, msize_vol));
   CUDA_CALL(hipMemset(pc, 0, msize_vol));
   CUDA_CALL(hipMallocManaged(&qp, msize_vol));
   CUDA_CALL(hipMemset(qp, 0, msize_vol));
   CUDA_CALL(hipMallocManaged(&qc, msize_vol));
   CUDA_CALL(hipMemset(qc, 0, msize_vol));
   CUDA_CALL(hipMallocManaged(&fatAbsorb, msize_vol));
   if (!fatAbsorb) //dev_fatAbsorb, fatAbsorb, msize_vol, hipMemcpyHostToDevice));
             CUDA_CALL(hipMemset(fatAbsorb, 0, msize_vol));

  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
  printf("GPU memory usage = %ld MiB\n", 21*msize_vol/1024/1024);

}


void CUDA_Finalize()
{

   extern float* vpz;
   extern float* vsv;
   extern float* epsilon;
   extern float* delta;
   extern float* phi;
   extern float* theta;
   extern float* ch1dxx;
   extern float* ch1dyy;
   extern float* ch1dzz;
   extern float* ch1dxy;
   extern float* ch1dyz;
   extern float* ch1dxz;
   extern float* v2px;
   extern float* v2pz;
   extern float* v2sz;
   extern float* v2pn;
   extern float* pp;
   extern float* pc;
   extern float* qp;
   extern float* qc;
   extern float* fatAbsorb;

   CUDA_CALL(hipFree(vpz));
   CUDA_CALL(hipFree(vsv));
   CUDA_CALL(hipFree(epsilon));
   CUDA_CALL(hipFree(delta));
   CUDA_CALL(hipFree(phi));
   CUDA_CALL(hipFree(theta));
   CUDA_CALL(hipFree(ch1dxx));
   CUDA_CALL(hipFree(ch1dyy));
   CUDA_CALL(hipFree(ch1dzz));
   CUDA_CALL(hipFree(ch1dxy));
   CUDA_CALL(hipFree(ch1dyz));
   CUDA_CALL(hipFree(ch1dxz));
   CUDA_CALL(hipFree(v2px));
   CUDA_CALL(hipFree(v2pz));
   CUDA_CALL(hipFree(v2sz));
   CUDA_CALL(hipFree(v2pn));
   CUDA_CALL(hipFree(pp));
   CUDA_CALL(hipFree(pc));
   CUDA_CALL(hipFree(qp));
   CUDA_CALL(hipFree(qc));
   CUDA_CALL(hipFree(fatAbsorb));

   printf("CUDA_Finalize: SUCCESS\n");
}



// void CUDA_Update_pointers(const int sx, const int sy, const int sz, float *pc)
// {
//    extern float* pc;
//    const size_t sxsysz=((size_t)sx*sy)*sz;
//    const size_t msize_vol=sxsysz*sizeof(float);
//    // if (pc) //pc, dev_pc, msize_vol, hipMemcpyDeviceToHost));
// }
