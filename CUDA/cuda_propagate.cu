#include "hip/hip_runtime.h"
#include "cuda_defines.h"
#include "cuda_propagate.h"
#include "../derivatives.h"
#include "../map.h"

extern int deviceCount;

#ifdef UNIFIED
  #include <hip/hip_runtime.h>
  #include <nvToolsExt.h>
#endif


__global__ void kernel_Propagate(const int sx, const int sy, const int sz, const int bord,
	       const float dx, const float dy, const float dz, const float dt, const int it,
	       float * restrict ch1dxx, float * restrict ch1dyy, float * restrict ch1dzz,
	       float * restrict ch1dxy, float * restrict ch1dyz, float * restrict ch1dxz,
	       float * restrict v2px, float * restrict v2pz, float * restrict v2sz, float * restrict v2pn,
	       float * restrict pp, float * restrict pc, float * restrict qp, float * restrict qc, const int dev,
			   const int start, const int end)
{
  const int ix=(blockIdx.x * blockDim.x + threadIdx.x);
  const int iy=(blockIdx.y * blockDim.y + threadIdx.y);



#define SAMPLE_PRE_LOOP
#include "../sample.h"
#undef SAMPLE_PRE_LOOP

    // solve both equations in all internal grid points,
    // including absortion zone
    for (int iz = start; iz < end; iz++) {

#define SAMPLE_LOOP
#include "../sample.h"
#undef SAMPLE_LOOP

    }
}

// Propagate: using Fletcher's equations, propagate waves one dt,
//            either forward or backward in time
void CUDA_Propagate(const int sx, const int sy, const int sz, const int bord,
	       const float dx, const float dy, const float dz, const float dt, const int it,
	       float * restrict ch1dxx, float * restrict ch1dyy, float * restrict ch1dzz,
	       float * restrict ch1dxy, float * restrict ch1dyz, float * restrict ch1dxz,
	       float * restrict v2px, float * restrict v2pz, float * restrict v2sz, float * restrict v2pn,
        #ifdef UNIFIED
	       float *pp, float *pc, float *qp, float *qc
        #else
         float * restrict pp, float * restrict pc, float * restrict qp, float * restrict qc
        #endif
        )

{
  static int print;
  #ifndef UNIFIED
   extern float* dev_vpz;
   extern float* dev_vsv;
   extern float* dev_epsilon;
   extern float* dev_delta;
   extern float* dev_phi;
   extern float* dev_theta;
   extern float* dev_ch1dxx;
   extern float* dev_ch1dyy;
   extern float* dev_ch1dzz;
   extern float* dev_ch1dxy;
   extern float* dev_ch1dyz;
   extern float* dev_ch1dxz;
   extern float* dev_v2px;
   extern float* dev_v2pz;
   extern float* dev_v2sz;
   extern float* dev_v2pn;
   extern float* dev_pp;
   extern float* dev_pc;
   extern float* dev_qp;
   extern float* dev_qc;
   extern float* dev_fatAbsorb;
  #endif


  dim3 threadsPerBlock(BSIZE_X, BSIZE_Y);
  #ifdef UNIFIED
  dim3 numBlocks(sx/threadsPerBlock.x, sy/threadsPerBlock.y);
  #else
  dim3 numBlocks(sx/threadsPerBlock.x, sy/threadsPerBlock.y);
  #endif

  if(!print){
    print = 1;
    printf("sx=%d sy=%d sz=%d\n", sx, sy, sz);
    printf("running kernel_Propagate with (%d,%d) blocks of (%d,%d) threads\n", numBlocks.x, numBlocks.y, threadsPerBlock.x, threadsPerBlock.y);
  }
  #ifdef UNIFIED
  int d;
	// Call kernel for shared cells between GPUs

	const int chunk = (sz - 2 * bord - 2) / 2;
	//const int sizeGhost1 = 4 * (ind(1,0,0)-ind(0,0,0)) * (ind(0,1,0)-ind(0,0,0));
	const int sizeGhost2 = (ind(0,0,1)-ind(0,0,0));

	// printf("foda-se: %d %d %d\n", sizeGhost1, sizeGhost2, ind(0,0,0));

	nvtxRangePushA("Shared Voxels");
	for(d = deviceCount - 2; d >= 0; d--){
			int start = bord + 1 + (d + 1) * chunk - 4;
			int end = start + 8;
			hipSetDevice(d);
			kernel_Propagate <<<numBlocks, threadsPerBlock>>> (  sx,   sy,   sz,   bord,
		         dx,   dy,   dz,   dt,   it,
		        ch1dxx,  ch1dyy,  ch1dzz,
		        ch1dxy,  ch1dyz,  ch1dxz,
		        v2px,  v2pz,  v2sz,  v2pn,
		        pp,  pc,  qp,  qc, d, start, end);

	      CUDA_CALL(hipGetLastError());
				hipMemPrefetchAsync(pc + sizeGhost2 * (start + 4), sizeGhost2 * 4 * sizeof(float), d+1);
				//hipMemPrefetchAsync(qc + sizeGhost2 * (start + 4), sizeGhost2 * 4 * sizeof(float), d+1);

				printf("Prefetch pc: %p until %p (size: %lu) to dev:%d\n", pc + sizeGhost2 * (start + 4), pc + sizeGhost2 * (start + 4) + sizeGhost2 * 4, sizeGhost2 * 4 * sizeof(float), d+1);
				//printf("Prefetch qc: %p until %p (size: %lu) to dev:%d\n", qc + sizeGhost2 * (start + 4), qc + sizeGhost2 * (start + 4) + sizeGhost2 * 4, sizeGhost2 * 4 * sizeof(float), d+1);

				CUDA_CALL(hipGetLastError());
	}



	for(d = deviceCount - 2; d >= 0; d--){
			int start = bord + 1 + (d + 1) * chunk - 4;
			int end = start + 8;
			hipSetDevice(d);
			CUDA_CALL(hipDeviceSynchronize());
			hipMemPrefetchAsync(pp + sizeGhost2 * (start), sizeGhost2 * 8 * sizeof(float), d+1);
			//hipMemPrefetchAsync(qp + sizeGhost2 * (start), sizeGhost2 * 8 * sizeof(float), d+1);
			printf("Prefetch pp: %p until %p (size: %lu) to dev:%d\n", pp + sizeGhost2 * (start), pp + sizeGhost2 * (start) + sizeGhost2 * 8, sizeGhost2 * 8 * sizeof(float), d+1);
			//printf("Prefetch qp: %p until %p (size: %lu) to dev:%d\n", qp + sizeGhost2 * (start), qp + sizeGhost2 * (start) + sizeGhost2 * 8, sizeGhost2 * 8 * sizeof(float), d+1);

	}

	nvtxRangePop();
	nvtxRangePushA("Exclusive");
  for(d = deviceCount - 1; d >= 0; d--){
		int start = bord + 1 + d * chunk + 4;
		int end = bord + 1 + (d + 1) * chunk - 4;
		if(d==0){
			start-=4;
		}
		if(d == (deviceCount - 1)){
			end+=4;
		}

    hipSetDevice(d);
    kernel_Propagate <<<numBlocks, threadsPerBlock>>> (  sx,   sy,   sz,   bord,
	         dx,   dy,   dz,   dt,   it,
	        ch1dxx,  ch1dyy,  ch1dzz,
	        ch1dxy,  ch1dyz,  ch1dxz,
	        v2px,  v2pz,  v2sz,  v2pn,
	        pp,  pc,  qp,  qc, d, start, end);

      CUDA_CALL(hipGetLastError());

  }

	for(d = deviceCount - 1; d >= 0; d--){
		hipSetDevice(d);
		CUDA_CALL(hipDeviceSynchronize());
	}

	nvtxRangePop();
  #else
  kernel_Propagate <<<numBlocks, threadsPerBlock>>> (  sx,   sy,   sz,   bord,
           dx,   dy,   dz,   dt,   it,
          dev_ch1dxx,  dev_ch1dyy,  dev_ch1dzz,
          dev_ch1dxy,  dev_ch1dyz,  dev_ch1dxz,
          dev_v2px,  dev_v2pz,  dev_v2sz,  dev_v2pn,
          dev_pp,  dev_pc,  dev_qp,  dev_qc);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());
  #endif

  #ifdef UNIFIED
    CUDA_SwapArrays(&pp, &pc, &qp, &qc);
  #else
    CUDA_SwapArrays(&dev_pp, &dev_pc, &dev_qp, &dev_qc);
  #endif
}

// swap array pointers on time forward array propagation
void CUDA_SwapArrays(float **pp, float **pc, float **qp, float **qc) {
  float *tmp;

  tmp=*pp;
  *pp=*pc;
  *pc=tmp;

  tmp=*qp;
  *qp=*qc;
  *qc=tmp;
}
